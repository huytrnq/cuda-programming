#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


/**
 * @brief Check the result of the calculation
 * 
 * @param hostRef The reference result on the host
 * @param gpuRef The result on the device
 * @param N The size of the arrays
 */
#define CHECK(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

/**
 * @brief Check the result of the calculation
 * 
 * @param hostRef The reference result on the host
 * @param gpuRef The result on the device
 * @param N The size of the arrays
 */
void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if (match) printf("Arrays match.\n\n");
}


/**
 * @brief Initialize the data on the host
 * 
 * @param ip The pointer to the data
 * @param size The size of the data
 */
void intialData(float *ip, int size) {
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

/**
 * @brief Sum the arrays on the host
 * 
 * @param A The first array
 * @param B The second array
 * @param C The result array
 * @param N The size of the arrays
 */
void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}

/**
 * @brief Sum the arrays on the device
 * 
 * @param A The first array
 * @param B The second array
 * @param C The result array
 * @param N The size of the arrays
 */
__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main(int argc, char **argv) {
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipSetDevice(dev);

    // set up data size of vectors
    int nElem = 32;
    printf("Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data at host side
    intialData(h_A, nElem);
    intialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    dim3 block(nElem);
    dim3 grid(nElem / block.x);

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C);
    printf("Execution configuration <<<%d, %d>>>\n", grid.x, block.x);

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());

    return 0;
}